#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../utils/hpc.h"

#define ASSERT_NO_ERR(exp) {cudaSafeCall((exp)); cudaCheckError();}

__global__ void kernel(matrix_t in, model_t mdl, matrix_t out, int* layer)
{
    const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= out->n)
        return;

    const float** w = (const float**) mdl->weights_list[*layer]->data;
    const float* b = (const float*) mdl->bias_list[*layer]->data;

    for (unsigned int bn=0; bn<in->m; bn++) { //for each batch
        float sum = b[index];
        for (unsigned int k=0; k<R; k++) {
            sum += w[index][k] * in->data[bn][index+k];
        }
        out->data[bn][index] = (*layer != mdl->num_layer-1) ? ACTIVATION(sum) : sum;
    }
}

matrix_t cuda1_forward_mlp(matrix_t input_batch, model_t model)
{
    matrix_t d_in = h2d_matrix(input_batch);
    model_t d_mdl = h2d_model(model);

    matrix_t h_out = new_matrix(input_batch->m, input_batch->n, ZERO);
    matrix_t d_out = h2d_matrix(h_out);
    free_matrix(h_out);
    
    #define BLKDIM 1024
    const unsigned num_block = (input_batch->n + BLKDIM-1)/BLKDIM;
    const unsigned num_thread = BLKDIM;
    
    double tstart = hpc_gettime();
    int *d_layer;
    ASSERT_NO_ERR(hipMalloc((void**)&d_layer, sizeof(int)));
    for (int layer=0; layer<model->num_layer; layer++) {
        ASSERT_NO_ERR(hipMemcpy(d_layer, &layer, sizeof(int), hipMemcpyHostToDevice));
        ASSERT_NO_ERR(hipMemcpy(&(d_out->n), &(model->bias_list[layer]->len), sizeof(int), hipMemcpyHostToDevice));
        kernel<<<num_block, num_thread>>>(d_in, d_mdl, d_out, d_layer);
        cudaCheckError();
        
        if (layer < model->num_layer-1) {
            matrix_t swap;
            swap = d_in;
            d_in = d_out;
            d_out = swap;
        }
    }
    ASSERT_NO_ERR(hipFree(d_layer));
    double tstop = hpc_gettime();
    printf("P1 kernel time elapsed = %f\n", tstop-tstart);
    h_out = d2h_matrix(d_out);

    device_free_matrix(d_in);
    device_free_model(d_mdl);
    device_free_matrix(d_out);
    return h_out;
}

vector_t h2d_vector(vector_t vec) {
    vector_t d_vec;
    ASSERT_NO_ERR(hipMalloc((void**)&d_vec, sizeof(vector_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_vec->len), &(vec->len), sizeof(int), hipMemcpyHostToDevice));
    float* d_data;
    ASSERT_NO_ERR(hipMalloc((void**)&d_data, vec->len*sizeof(float)));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_vec->data, &d_data, sizeof(float*), hipMemcpyHostToDevice));
    return d_vec;
}

vector_t d2h_vector(vector_t d_vec) {
    vector_t vec = (vector_t) malloc(sizeof(vector_s));
    ASSERT_NO_ERR(hipMemcpy(vec, d_vec, sizeof(vector_s), hipMemcpyDeviceToHost));
    float* d_data = (float*) malloc(vec->len*sizeof(float));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyDeviceToHost));
    vec->data = d_data;
    return vec;
}

void device_free_vector(vector_t d_vec) {
    float* d_data;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_vec->data, sizeof(float*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_vec));
}

matrix_t h2d_matrix(matrix_t mat) {
    matrix_t d_mat;
    ASSERT_NO_ERR(hipMalloc((void**)&d_mat, sizeof(matrix_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->m), &(mat->m), sizeof(unsigned int), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->n), &(mat->n), sizeof(unsigned int), hipMemcpyHostToDevice));
    float** d_data;
    float* d_blk;
    ASSERT_NO_ERR(hipMalloc((void**)&d_data, mat->m*sizeof(float*)));
    ASSERT_NO_ERR(hipMalloc((void**)&d_blk, mat->m*mat->n*sizeof(float)));
    ASSERT_NO_ERR(hipMemcpy(d_blk, mat->data[0], mat->m*mat->n*sizeof(float), hipMemcpyHostToDevice));
    float* addr[mat->m];
    for (unsigned int i=0; i<mat->m; i++) {
        addr[i] = &d_blk[i*mat->n];
    }
    ASSERT_NO_ERR(hipMemcpy(d_data, addr, mat->m*sizeof(float*), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_mat->data, &d_data, sizeof(float**), hipMemcpyHostToDevice));
    return d_mat;
}

matrix_t d2h_matrix(matrix_t d_mat) {
    matrix_t mat = (matrix_t) malloc(sizeof(matrix_s));
    ASSERT_NO_ERR(hipMemcpy(mat, d_mat, sizeof(matrix_s), hipMemcpyDeviceToHost));
    float** data = (float**) malloc(mat->m*sizeof(float*));
    ASSERT_NO_ERR(hipMemcpy(data, mat->data, mat->m*sizeof(float*), hipMemcpyDeviceToHost));
    float* blk = (float*) malloc(mat->m*mat->n*sizeof(float));
    ASSERT_NO_ERR(hipMemcpy(blk, data[0], mat->m*mat->n*sizeof(float), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<mat->m; i++) {
        data[i] = &(blk[i*mat->n]);
    }
    mat->data = data;
    return mat;
}

void device_free_matrix(matrix_t d_mat) {
    float** d_data;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_mat->data, sizeof(float**), hipMemcpyDeviceToHost));
    float *d_blk;
    ASSERT_NO_ERR(hipMemcpy(&d_blk, d_data, sizeof(float*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipFree(d_blk));
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_mat));
}

model_t h2d_model(model_t mdl) {
    model_t d_mdl;
    ASSERT_NO_ERR(hipMalloc((void**)&d_mdl, sizeof(model_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_mdl->num_layer), &(mdl->num_layer), sizeof(unsigned int), hipMemcpyHostToDevice));
    matrix_t* d_weights;
    vector_t* d_biases;
    ASSERT_NO_ERR(hipMalloc((void**)&d_weights, mdl->num_layer*sizeof(matrix_t*)));
    ASSERT_NO_ERR(hipMalloc((void**)&d_biases, mdl->num_layer*sizeof(vector_t*)));
    for (unsigned int i=0; i<mdl->num_layer; i++) {
        matrix_t d_m = h2d_matrix(mdl->weights_list[i]);
        vector_t d_v = h2d_vector(mdl->bias_list[i]);
        ASSERT_NO_ERR(hipMemcpy(&d_weights[i], &d_m, sizeof(matrix_t), hipMemcpyHostToDevice));
        ASSERT_NO_ERR(hipMemcpy(&d_biases[i], &d_v, sizeof(vector_t), hipMemcpyHostToDevice));
    }
    ASSERT_NO_ERR(hipMemcpy(&d_mdl->weights_list, &d_weights, sizeof(matrix_t*), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_mdl->bias_list, &d_biases, sizeof(vector_t*), hipMemcpyHostToDevice));
    return d_mdl;
}

void device_free_model(model_t d_mdl) {
    matrix_t* d_weights;
    vector_t* d_biases;
    unsigned int num_layer;
    ASSERT_NO_ERR(hipMemcpy(&num_layer, &d_mdl->num_layer, sizeof(unsigned int), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&d_weights, &d_mdl->weights_list, sizeof(matrix_t*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&d_biases, &d_mdl->bias_list, sizeof(vector_t*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<num_layer; i++) {
        matrix_t d_w;
        vector_t d_b;
        ASSERT_NO_ERR(hipMemcpy(&d_w, &d_weights[i], sizeof(matrix_t), hipMemcpyDeviceToHost));
        ASSERT_NO_ERR(hipMemcpy(&d_b, &d_biases[i], sizeof(vector_t), hipMemcpyDeviceToHost));
        
        device_free_matrix(d_w);
        device_free_vector(d_b);
    }
    ASSERT_NO_ERR(hipFree(d_weights));
    ASSERT_NO_ERR(hipFree(d_biases));
    ASSERT_NO_ERR(hipFree(d_mdl));
}

void test_device_mem_leak() {
    const unsigned int cycles = 500;
    vector_t vec = new_vector(1000, RAND_UNIFORM);
    matrix_t mat = new_matrix(1000, 1000, RAND_UNIFORM);
    model_t mdl = new_model(100, 20, RAND_UNIFORM);
    printf("Device memory leak stress test: running %d cycles...\n", cycles);

    for (unsigned int i=0; i<cycles; i++) {
        vector_t d_vec = h2d_vector(vec);
        vector_t h_vec = d2h_vector(d_vec);
        assert_equal_vector(vec, h_vec);
        free_vector(h_vec);
        device_free_vector(d_vec);

        matrix_t d_mat = h2d_matrix(mat);
        matrix_t h_mat = d2h_matrix(d_mat);
        assert_equal_matrix(mat, h_mat);
        free_matrix(h_mat);
        device_free_matrix(d_mat);
        
        model_t d_mdl = h2d_model(mdl);
        device_free_model(d_mdl);
        printf("%d  ", i); fflush(stdout);
    }
    printf("\n");
}