#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../utils/hpc.h"

#define ASSERT_NO_ERR(exp) {cudaSafeCall((exp)); cudaCheckError();}

__global__ void add( int *a, int *b, int *c )
{
    *c = *a + *b;
}

matrix_t cuda1_forward_mlp(matrix_t input_batch, model_t model)
{
    for(int i=0; i<10000; i++) {
        vector_t d_vec = h2d_vector(model->bias_list[0]);
        vector_t h_vec = d2h_vector(d_vec);
        print_vector(model->bias_list[0]);
        print_vector(h_vec);
        free_vector(h_vec);
        device_free_vector(d_vec);
        printf("vec OK\n");

        matrix_t d_mat = h2d_matrix(model->weights_list[0]);
        matrix_t h_mat = d2h_matrix(d_mat);
        print_matrix(model->weights_list[0]);
        print_matrix(h_mat);
        free_matrix(h_mat);
        device_free_matrix(d_mat);
        printf("mat OK\n");

        model_t d_mdl = h2d_model(model);
        device_free_model(d_mdl);
        printf("mdl OK\n");
    }
}

vector_t h2d_vector(vector_t vec) {
    vector_t d_vec;
    ASSERT_NO_ERR(hipMalloc(&d_vec, sizeof(vector_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_vec->len), &(vec->len), sizeof(int), hipMemcpyHostToDevice));
    float* d_data;
    ASSERT_NO_ERR(hipMalloc(&d_data, vec->len*sizeof(float)));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_vec->data, &d_data, sizeof(float*), hipMemcpyHostToDevice));
    return d_vec;
}

vector_t d2h_vector(vector_t d_vec) {
    vector_t vec = (vector_t) malloc(sizeof(vector_s));
    ASSERT_NO_ERR(hipMemcpy(vec, d_vec, sizeof(vector_s), hipMemcpyDeviceToHost));
    float* d_data = (float*) malloc(vec->len*sizeof(float));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyDeviceToHost));
    vec->data = d_data;
    return vec;
}

void device_free_vector(vector_t d_vec) {
    float* d_data;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_vec->data, sizeof(float*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_vec));
}

matrix_t h2d_matrix(matrix_t mat) {
    matrix_t d_mat;
    ASSERT_NO_ERR(hipMalloc(&d_mat, sizeof(matrix_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->m), &(mat->m), sizeof(unsigned int), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->n), &(mat->n), sizeof(unsigned int), hipMemcpyHostToDevice));
    float** d_data;
    ASSERT_NO_ERR(hipMalloc(&d_data, mat->m*sizeof(float*)));
    for (unsigned int i=0; i<mat->m; i++) {
        float *d_row;
        ASSERT_NO_ERR(hipMalloc(&d_row, mat->n*sizeof(float)));
        ASSERT_NO_ERR(hipMemcpy(d_row, mat->data[i], mat->n*sizeof(float), hipMemcpyHostToDevice));
        ASSERT_NO_ERR(hipMemcpy(&d_data[i], &d_row, sizeof(float*), hipMemcpyHostToDevice));
    }
    ASSERT_NO_ERR(hipMemcpy(&d_mat->data, &d_data, sizeof(float**), hipMemcpyHostToDevice));
    return d_mat;
}

matrix_t d2h_matrix(matrix_t d_mat) {
    matrix_t mat = (matrix_t) malloc(sizeof(matrix_s));
    ASSERT_NO_ERR(hipMemcpy(mat, d_mat, sizeof(matrix_s), hipMemcpyDeviceToHost));
    float** d_data = (float**) malloc(mat->m*sizeof(float*));
    ASSERT_NO_ERR(hipMemcpy(d_data, mat->data, mat->m*sizeof(float*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<mat->m; i++) {
        float* d_row = (float*) malloc(mat->n*sizeof(float));
        ASSERT_NO_ERR(hipMemcpy(d_row, d_data[i], mat->n*sizeof(float), hipMemcpyDeviceToHost));
        d_data[i] = d_row;
    }
    mat->data = d_data;
    return mat;
}

void device_free_matrix(matrix_t d_mat) {
    float** d_data;
    unsigned int m;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_mat->data, sizeof(float**), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&m, &d_mat->m, sizeof(unsigned int), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<m; i++) {
        float* d_row;
        ASSERT_NO_ERR(hipMemcpy(&d_row, &d_data[i], sizeof(float*), hipMemcpyDeviceToHost));
        ASSERT_NO_ERR(hipFree(d_row));
    }
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_mat));
}

model_t h2d_model(model_t mdl) {
    model_t d_mdl;
    ASSERT_NO_ERR(hipMalloc(&d_mdl, sizeof(model_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_mdl->num_layer), &(mdl->num_layer), sizeof(unsigned int), hipMemcpyHostToDevice));
    matrix_t* d_weights;
    vector_t* d_biases;
    ASSERT_NO_ERR(hipMalloc(&d_weights, mdl->num_layer*sizeof(matrix_t*)));
    ASSERT_NO_ERR(hipMalloc(&d_biases, mdl->num_layer*sizeof(vector_t*)));
    for (unsigned int i=0; i<mdl->num_layer; i++) {
        matrix_t d_m = h2d_matrix(mdl->weights_list[i]);
        vector_t d_v = h2d_vector(mdl->bias_list[i]);
        ASSERT_NO_ERR(hipMemcpy(&d_weights[i], &d_m, sizeof(matrix_t), hipMemcpyHostToDevice));
        ASSERT_NO_ERR(hipMemcpy(&d_biases[i], &d_v, sizeof(vector_t), hipMemcpyHostToDevice));
    }
    ASSERT_NO_ERR(hipMemcpy(&d_mdl->weights_list, &d_weights, sizeof(matrix_t*), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_mdl->bias_list, &d_biases, sizeof(vector_t*), hipMemcpyHostToDevice));
    return d_mdl;
}

void device_free_model(model_t d_mdl) {
    matrix_t* d_weights;
    vector_t* d_biases;
    unsigned int num_layer;
    ASSERT_NO_ERR(hipMemcpy(&num_layer, &d_mdl->num_layer, sizeof(unsigned int), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&d_weights, &d_mdl->weights_list, sizeof(matrix_t*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&d_biases, &d_mdl->bias_list, sizeof(vector_t*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<num_layer; i++) {
        matrix_t d_w;
        vector_t d_b;
        ASSERT_NO_ERR(hipMemcpy(&d_w, &d_weights[i], sizeof(matrix_t), hipMemcpyDeviceToHost));
        ASSERT_NO_ERR(hipMemcpy(&d_b, &d_biases[i], sizeof(vector_t), hipMemcpyDeviceToHost));
        device_free_matrix(d_w);
        device_free_vector(d_b);
    }
    ASSERT_NO_ERR(hipFree(d_weights));
    ASSERT_NO_ERR(hipFree(d_biases));
    ASSERT_NO_ERR(hipFree(d_mdl));
}