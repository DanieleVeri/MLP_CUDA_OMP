#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../utils/hpc.h"

#define BLKDIM 1024

__global__ void kernel_layer(matrix_t in, model_t mdl, matrix_t out, int* layer)
{
    __shared__ float temp_i[BLKDIM+R];
    const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= out->n)
        return;

    const float** w = (const float**) mdl->weights_list[*layer]->data;
    const float* b = (const float*) mdl->bias_list[*layer]->data;

    temp_i[threadIdx.x] = in->data[0][index];
    if (threadIdx.x < R) {
        int missing = out->n - index;
        if (missing < BLKDIM)
            temp_i[2*threadIdx.x+missing] = in->data[0][threadIdx.x+out->n];
        else
            temp_i[threadIdx.x+blockDim.x] = in->data[0][index+blockDim.x];
    }
    __syncthreads();

    for (unsigned int bn=0; bn<in->m; bn++) { //for each batch
        float sum = b[index];
        for (unsigned int k=0; k<R; k++) {
            sum += w[index][k] * temp_i[threadIdx.x+k];
        }
        out->data[bn][index] = (*layer != mdl->num_layer-1) ? ACTIVATION(sum) : sum;
    }
}

__host__ matrix_t cuda1_forward_mlp(matrix_t input_batch, model_t model)
{
    matrix_t d_in = h2d_matrix(input_batch);
    model_t d_mdl = h2d_model(model);

    matrix_t h_out = new_matrix(input_batch->m, input_batch->n, ZERO);
    matrix_t d_out = h2d_matrix(h_out);
    free_matrix(h_out);
    
    const unsigned num_block = (input_batch->n + BLKDIM-1)/BLKDIM;
    const unsigned num_thread = BLKDIM;
    
    double tstart = hpc_gettime();

    int *d_layer;
    cudaSafeCall(hipMalloc((void**)&d_layer, sizeof(int)));

    for (int layer=0; layer<model->num_layer; layer++) {
        cudaSafeCall(hipMemcpy(d_layer, &layer, sizeof(int), hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy(&(d_out->n), &(model->bias_list[layer]->len), sizeof(int), hipMemcpyHostToDevice));

        kernel_layer<<<num_block, num_thread>>>(d_in, d_mdl, d_out, d_layer);
        cudaCheckError();
        
        if (layer < model->num_layer-1) {
            matrix_t swap;
            swap = d_in;
            d_in = d_out;
            d_out = swap;
        }
    }

    cudaSafeCall(hipFree(d_layer));

    double tstop = hpc_gettime();
    printf("P1 kernel time elapsed = %f\n", tstop-tstart);
    h_out = d2h_matrix(d_out);

    device_free_matrix(d_in);
    device_free_model(d_mdl);
    device_free_matrix(d_out);
    return h_out;
}

__host__ vector_t h2d_vector(vector_t vec) {
    vector_t d_vec;
    cudaSafeCall(hipMalloc((void**)&d_vec, sizeof(vector_s)));
    cudaSafeCall(hipMemcpy(&(d_vec->len), &(vec->len), sizeof(int), hipMemcpyHostToDevice));
    float* d_data;
    cudaSafeCall(hipMalloc((void**)&d_data, vec->len*sizeof(float)));
    cudaSafeCall(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_vec->data, &d_data, sizeof(float*), hipMemcpyHostToDevice));
    return d_vec;
}

__host__ vector_t d2h_vector(vector_t d_vec) {
    vector_t vec = (vector_t) malloc(sizeof(vector_s));
    cudaSafeCall(hipMemcpy(vec, d_vec, sizeof(vector_s), hipMemcpyDeviceToHost));
    float* d_data = (float*) malloc(vec->len*sizeof(float));
    cudaSafeCall(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyDeviceToHost));
    vec->data = d_data;
    return vec;
}

__host__ void device_free_vector(vector_t d_vec) {
    float* d_data;
    cudaSafeCall(hipMemcpy(&d_data, &d_vec->data, sizeof(float*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipFree(d_vec));
}

__host__ matrix_t h2d_matrix(matrix_t mat) {
    matrix_t d_mat;
    cudaSafeCall(hipMalloc((void**)&d_mat, sizeof(matrix_s)));
    cudaSafeCall(hipMemcpy(&(d_mat->m), &(mat->m), sizeof(unsigned int), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&(d_mat->n), &(mat->n), sizeof(unsigned int), hipMemcpyHostToDevice));
    float** d_data;
    float* d_blk;
    cudaSafeCall(hipMalloc((void**)&d_data, mat->m*sizeof(float*)));
    cudaSafeCall(hipMalloc((void**)&d_blk, mat->m*mat->n*sizeof(float)));
    cudaSafeCall(hipMemcpy(d_blk, mat->data[0], mat->m*mat->n*sizeof(float), hipMemcpyHostToDevice));
    float* addr[mat->m];
    for (unsigned int i=0; i<mat->m; i++) {
        addr[i] = &d_blk[i*mat->n];
    }
    cudaSafeCall(hipMemcpy(d_data, addr, mat->m*sizeof(float*), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_mat->data, &d_data, sizeof(float**), hipMemcpyHostToDevice));
    return d_mat;
}

__host__ matrix_t d2h_matrix(matrix_t d_mat) {
    matrix_t mat = (matrix_t) malloc(sizeof(matrix_s));
    cudaSafeCall(hipMemcpy(mat, d_mat, sizeof(matrix_s), hipMemcpyDeviceToHost));
    float** data = (float**) malloc(mat->m*sizeof(float*));
    cudaSafeCall(hipMemcpy(data, mat->data, mat->m*sizeof(float*), hipMemcpyDeviceToHost));
    float* blk = (float*) malloc(mat->m*mat->n*sizeof(float));
    cudaSafeCall(hipMemcpy(blk, data[0], mat->m*mat->n*sizeof(float), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<mat->m; i++) {
        data[i] = &(blk[i*mat->n]);
    }
    mat->data = data;
    return mat;
}

__host__ void device_free_matrix(matrix_t d_mat) {
    float** d_data;
    cudaSafeCall(hipMemcpy(&d_data, &d_mat->data, sizeof(float**), hipMemcpyDeviceToHost));
    float *d_blk;
    cudaSafeCall(hipMemcpy(&d_blk, d_data, sizeof(float*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(d_blk));
    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipFree(d_mat));
}

__host__ model_t h2d_model(model_t mdl) {
    model_t d_mdl;
    cudaSafeCall(hipMalloc((void**)&d_mdl, sizeof(model_s)));
    cudaSafeCall(hipMemcpy(&(d_mdl->num_layer), &(mdl->num_layer), sizeof(unsigned int), hipMemcpyHostToDevice));
    matrix_t* d_weights;
    vector_t* d_biases;
    cudaSafeCall(hipMalloc((void**)&d_weights, mdl->num_layer*sizeof(matrix_t*)));
    cudaSafeCall(hipMalloc((void**)&d_biases, mdl->num_layer*sizeof(vector_t*)));
    for (unsigned int i=0; i<mdl->num_layer; i++) {
        matrix_t d_m = h2d_matrix(mdl->weights_list[i]);
        vector_t d_v = h2d_vector(mdl->bias_list[i]);
        cudaSafeCall(hipMemcpy(&d_weights[i], &d_m, sizeof(matrix_t), hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy(&d_biases[i], &d_v, sizeof(vector_t), hipMemcpyHostToDevice));
    }
    cudaSafeCall(hipMemcpy(&d_mdl->weights_list, &d_weights, sizeof(matrix_t*), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_mdl->bias_list, &d_biases, sizeof(vector_t*), hipMemcpyHostToDevice));
    return d_mdl;
}

__host__ void device_free_model(model_t d_mdl) {
    matrix_t* d_weights;
    vector_t* d_biases;
    unsigned int num_layer;
    cudaSafeCall(hipMemcpy(&num_layer, &d_mdl->num_layer, sizeof(unsigned int), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(&d_weights, &d_mdl->weights_list, sizeof(matrix_t*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(&d_biases, &d_mdl->bias_list, sizeof(vector_t*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<num_layer; i++) {
        matrix_t d_w;
        vector_t d_b;
        cudaSafeCall(hipMemcpy(&d_w, &d_weights[i], sizeof(matrix_t), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&d_b, &d_biases[i], sizeof(vector_t), hipMemcpyDeviceToHost));
        
        device_free_matrix(d_w);
        device_free_vector(d_b);
    }
    cudaSafeCall(hipFree(d_weights));
    cudaSafeCall(hipFree(d_biases));
    cudaSafeCall(hipFree(d_mdl));
}

__host__ void test_device_mem_leak() {
    const unsigned int cycles = 500;
    vector_t vec = new_vector(1000, RAND_UNIFORM);
    matrix_t mat = new_matrix(1000, 1000, RAND_UNIFORM);
    model_t mdl = new_model(100, 20, RAND_UNIFORM);
    printf("Device memory leak stress test: running %d cycles...\n", cycles);

    for (unsigned int i=0; i<cycles; i++) {
        vector_t d_vec = h2d_vector(vec);
        vector_t h_vec = d2h_vector(d_vec);
        assert_equal_vector(vec, h_vec);
        free_vector(h_vec);
        device_free_vector(d_vec);

        matrix_t d_mat = h2d_matrix(mat);
        matrix_t h_mat = d2h_matrix(d_mat);
        assert_equal_matrix(mat, h_mat);
        free_matrix(h_mat);
        device_free_matrix(d_mat);
        
        model_t d_mdl = h2d_model(mdl);
        device_free_model(d_mdl);
        printf("%d  ", i); fflush(stdout);
    }
    printf("\n");
}