#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../utils/hpc.h"

#define BLKDIM 1024

__global__ void kernel_layer(matrix_t in, model_t mdl, matrix_t out, int* layer)
{
    // shared memory to exploit data reuse of the input
    __shared__ float temp_i[BLKDIM+R];
    // index and batch
    const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int batch = blockIdx.y;
    // current layer weights and biases
    const float** w = (const float**) mdl->weights_list[*layer]->data;
    const float* b = (const float*) mdl->bias_list[*layer]->data;
    // current output size
    const unsigned max_n = mdl->bias_list[*layer]->len;
    if (index >= max_n)
        return;

    // fill shared mem handling the case that input is not a multiple of block size
    temp_i[threadIdx.x] = in->data[batch][index];
    if (threadIdx.x < R) {
        int missing = max_n - index;
        if (missing < BLKDIM)
            temp_i[2*threadIdx.x+missing] = in->data[batch][threadIdx.x+max_n];
        else
            temp_i[threadIdx.x+blockDim.x] = in->data[batch][index+blockDim.x];
    }
    __syncthreads();

    // compute neuron output
    float sum = b[index];
    for (unsigned int k=0; k<R; k++) {
        sum += w[index][k] * temp_i[threadIdx.x+k];
    }
    // apply activation if isn't the last layer (the model is a regressor)
    out->data[batch][index] = (*layer != mdl->num_layer-1) ? ACTIVATION(sum) : sum;
}

__host__ matrix_t cuda_forward_mlp(matrix_t input_batch, model_t model)
{
    // alloc and copy input and model to device memory
    matrix_t d_in = h2d_matrix(input_batch);
    model_t d_mdl = h2d_model(model);

    // create buffer matrix as big as the input (create on host, move on device, free on host)
    matrix_t h_buff = new_matrix_pinned(input_batch->m, input_batch->n, ZERO);
    matrix_t d_buff = h2d_matrix(h_buff);
    free_matrix_pinned(h_buff);
    
    // 2D grid, x is for number of features, y is for batch elements
    dim3 grid((input_batch->n + BLKDIM-1)/BLKDIM, input_batch->m);
    dim3 block(BLKDIM);
    
    // Alloc on device current layer index
    int *d_layer;  cudaSafeCall(hipMalloc((void**)&d_layer, sizeof(int)));
    double tstart = hpc_gettime();
    // layer loop
    for (int layer=0; layer<model->num_layer; layer++) {
        // update current layer index on device
        cudaSafeCall(hipMemcpy(d_layer, &layer, sizeof(int), hipMemcpyHostToDevice));

        kernel_layer<<<grid, block>>>(d_in, d_mdl, d_buff, d_layer);
        hipDeviceSynchronize();
        cudaCheckError();
        
        // swap input and output buffer pointers
        if (layer < model->num_layer-1) {
            matrix_t swap;
            swap = d_in;
            d_in = d_buff;
            d_buff = swap;
        }
    }
    double tstop = hpc_gettime();
    printf("Kernel time elapsed = %f s\n", tstop-tstart);
    cudaSafeCall(hipFree(d_layer));

    // copy the result on host memory
    matrix_t h_out = d2h_matrix(d_buff);
    // truncate to the output dimension (it was as big as the input)
    h_out->n = model->bias_list[model->num_layer-1]->len;

    device_free_matrix(d_in);
    device_free_model(d_mdl);
    device_free_matrix(d_buff);
    return h_out;
}

/********************************** VECTOR MEMORY UTILITIES **********************************/

__host__ vector_t new_vector_pinned(unsigned int length, init_t init_type) 
{
    vector_t vec;
    cudaSafeCall(hipHostMalloc(&vec, sizeof(vector_s)));
    vec->len = length;
    cudaSafeCall(hipHostMalloc(&vec->data, length * sizeof(float)));
    for (unsigned int i=0; i<length; i++) {
        switch (init_type)
        {
        case RAND_UNIFORM: // [-2, +2)
            vec->data[i] = 4 * ((float) rand() / (float) RAND_MAX - 0.5);
            break;
        default:
            vec->data[i] = 0;
            break;
        }
        
    }
    return vec;
}

__host__ void free_vector_pinned(vector_t vector) 
{
    cudaSafeCall(hipHostFree(vector->data));
    cudaSafeCall(hipHostFree(vector));
}

__host__ vector_t h2d_vector(vector_t vec) {
    // alloc the struct
    vector_t d_vec;
    cudaSafeCall(hipMalloc((void**)&d_vec, sizeof(vector_s)));
    cudaSafeCall(hipMemcpy(&(d_vec->len), &(vec->len), sizeof(int), hipMemcpyHostToDevice));
    // alloc the data, then update the vec->data pointer with that one on the device memory
    float* d_data;
    cudaSafeCall(hipMalloc((void**)&d_data, vec->len*sizeof(float)));
    cudaSafeCall(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_vec->data, &d_data, sizeof(float*), hipMemcpyHostToDevice));
    return d_vec;
}

__host__ vector_t d2h_vector(vector_t d_vec) {
    // copy on host the struct
    vector_t vec;
    cudaSafeCall(hipHostMalloc(&vec, sizeof(vector_s)));
    cudaSafeCall(hipMemcpy(vec, d_vec, sizeof(vector_s), hipMemcpyDeviceToHost));
    // copy on host the data
    float* d_data;
    cudaSafeCall(hipHostMalloc(&d_data, vec->len*sizeof(float)));
    cudaSafeCall(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyDeviceToHost));
    vec->data = d_data;
    return vec;
}

__host__ void device_free_vector(vector_t d_vec) {
    float* d_data;
    cudaSafeCall(hipMemcpy(&d_data, &d_vec->data, sizeof(float*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipFree(d_vec));
}

/********************************** MATRIX MEMORY UTILITIES **********************************/

__host__ matrix_t new_matrix_pinned(unsigned int m, unsigned int n, init_t init_type) 
{
    matrix_t mat;
    cudaSafeCall(hipHostMalloc(&mat, sizeof(vector_s)));
    mat->m = m; mat->n = n;
    // data block (contiguous)
    float* blk;
    cudaSafeCall(hipHostMalloc(&blk, m * n * sizeof(float)));
    cudaSafeCall(hipHostMalloc(&mat->data, m * sizeof(float*)));
    // updates the elements of data with the addresses of data block
    // allowing to preserve the double indexing of the array
    for (unsigned int i=0; i<m; i++) {
        mat->data[i] = &(blk[i*n]);
        for (unsigned int j=0; j<n; j++) {
            switch (init_type)
            {
            case RAND_UNIFORM: // [-2, +2)
                mat->data[i][j] = 4 * ((float) rand() / (float) RAND_MAX - 0.5);
                break;
            default:
                mat->data[i][j] = 0;
                break;
            }
        }
    }
    return mat;
}

__host__ void free_matrix_pinned(matrix_t matrix)
{
    cudaSafeCall(hipHostFree(matrix->data[0])); //free blk
    cudaSafeCall(hipHostFree(matrix->data));
    cudaSafeCall(hipHostFree(matrix));
}

__host__ matrix_t h2d_matrix(matrix_t mat) {
    // copy on device the struct
    matrix_t d_mat;
    cudaSafeCall(hipMalloc((void**)&d_mat, sizeof(matrix_s)));
    cudaSafeCall(hipMemcpy(&(d_mat->m), &(mat->m), sizeof(unsigned int), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&(d_mat->n), &(mat->n), sizeof(unsigned int), hipMemcpyHostToDevice));
    // copy on device the data (d_blk)
    float** d_data;
    float* d_blk;
    cudaSafeCall(hipMalloc((void**)&d_data, mat->m*sizeof(float*)));
    cudaSafeCall(hipMalloc((void**)&d_blk, mat->m*mat->n*sizeof(float)));
    cudaSafeCall(hipMemcpy(d_blk, mat->data[0], mat->m*mat->n*sizeof(float), hipMemcpyHostToDevice));
    // update the d_data indeces with that ones od d_blk on the device memory, so to preserve the double index access
    float* addr[mat->m];
    for (unsigned int i=0; i<mat->m; i++) {
        addr[i] = &d_blk[i*mat->n];
    }
    cudaSafeCall(hipMemcpy(d_data, addr, mat->m*sizeof(float*), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_mat->data, &d_data, sizeof(float**), hipMemcpyHostToDevice));
    return d_mat;
}

__host__ matrix_t d2h_matrix(matrix_t d_mat) {
    // copy struct to host
    matrix_t mat;
    cudaSafeCall(hipHostMalloc(&mat, sizeof(matrix_s)));
    cudaSafeCall(hipMemcpy(mat, d_mat, sizeof(matrix_s), hipMemcpyDeviceToHost));
    // copy data to host (data[0] will contain the memory block address)
    float** data;
    cudaSafeCall(hipHostMalloc(&data, mat->m*sizeof(float*)));
    cudaSafeCall(hipMemcpy(data, mat->data, mat->m*sizeof(float*), hipMemcpyDeviceToHost));
    // copy the memory block to host
    float* blk;
    cudaSafeCall(hipHostMalloc(&blk, mat->m*mat->n*sizeof(float)));
    cudaSafeCall(hipMemcpy(blk, data[0], mat->m*mat->n*sizeof(float), hipMemcpyDeviceToHost));
    // update data indeces with the host copy of blk
    for (unsigned int i=0; i<mat->m; i++) {
        data[i] = &(blk[i*mat->n]);
    }
    mat->data = data;
    return mat;
}

__host__ void device_free_matrix(matrix_t d_mat) {
    float** d_data;
    cudaSafeCall(hipMemcpy(&d_data, &d_mat->data, sizeof(float**), hipMemcpyDeviceToHost));
    float *d_blk;
    cudaSafeCall(hipMemcpy(&d_blk, d_data, sizeof(float*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipFree(d_blk));
    cudaSafeCall(hipFree(d_data));
    cudaSafeCall(hipFree(d_mat));
}

/********************************** MODEL MEMORY UTILITIES **********************************/

__host__ model_t new_model_pinned(unsigned int inputs, unsigned int num_layer, init_t init_type) {
    model_t obj;
    cudaSafeCall(hipHostMalloc(&obj, sizeof(model_s)));
    obj->num_layer = num_layer;
    cudaSafeCall(hipHostMalloc(&obj->weights_list, num_layer * sizeof(matrix_t)));
    cudaSafeCall(hipHostMalloc(&obj->bias_list, num_layer * sizeof(vector_t)));
    unsigned int last = inputs;
    for (unsigned int i=0; i<num_layer; i++) {
        obj->weights_list[i] = new_matrix_pinned(last-(R-1), R, init_type);
        obj->bias_list[i] = new_vector_pinned(last-(R-1), init_type);
        last -= R-1;
    }
    return obj;
}

__host__ void free_model_pinned(model_t model) {
    for (unsigned int i=0; i<model->num_layer; i++) {
        free_matrix_pinned(model->weights_list[i]);
        free_vector_pinned(model->bias_list[i]);
    }
    cudaSafeCall(hipHostFree(model->weights_list));
    cudaSafeCall(hipHostFree(model->bias_list));
    cudaSafeCall(hipHostFree(model));
}

__host__ model_t h2d_model(model_t mdl) {
    // copy struct to device
    model_t d_mdl;
    cudaSafeCall(hipMalloc((void**)&d_mdl, sizeof(model_s)));
    cudaSafeCall(hipMemcpy(&(d_mdl->num_layer), &(mdl->num_layer), sizeof(unsigned int), hipMemcpyHostToDevice));
    // alloc the pointer lists
    matrix_t* d_weights;
    vector_t* d_biases;
    cudaSafeCall(hipMalloc((void**)&d_weights, mdl->num_layer*sizeof(matrix_t*)));
    cudaSafeCall(hipMalloc((void**)&d_biases, mdl->num_layer*sizeof(vector_t*)));
    // copy all weights and biases and update the lists with device pointers
    for (unsigned int i=0; i<mdl->num_layer; i++) {
        matrix_t d_m = h2d_matrix(mdl->weights_list[i]);
        vector_t d_v = h2d_vector(mdl->bias_list[i]);
        cudaSafeCall(hipMemcpy(&d_weights[i], &d_m, sizeof(matrix_t), hipMemcpyHostToDevice));
        cudaSafeCall(hipMemcpy(&d_biases[i], &d_v, sizeof(vector_t), hipMemcpyHostToDevice));
    }
    cudaSafeCall(hipMemcpy(&d_mdl->weights_list, &d_weights, sizeof(matrix_t*), hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(&d_mdl->bias_list, &d_biases, sizeof(vector_t*), hipMemcpyHostToDevice));
    return d_mdl;
}

__host__ void device_free_model(model_t d_mdl) {
    // copy to host the pointer lists, than free each element
    matrix_t* d_weights;
    vector_t* d_biases;
    unsigned int num_layer;
    cudaSafeCall(hipMemcpy(&num_layer, &d_mdl->num_layer, sizeof(unsigned int), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(&d_weights, &d_mdl->weights_list, sizeof(matrix_t*), hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(&d_biases, &d_mdl->bias_list, sizeof(vector_t*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<num_layer; i++) {
        matrix_t d_w;
        vector_t d_b;
        cudaSafeCall(hipMemcpy(&d_w, &d_weights[i], sizeof(matrix_t), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&d_b, &d_biases[i], sizeof(vector_t), hipMemcpyDeviceToHost));
        device_free_matrix(d_w);
        device_free_vector(d_b);
    }
    cudaSafeCall(hipFree(d_weights));
    cudaSafeCall(hipFree(d_biases));
    cudaSafeCall(hipFree(d_mdl));
}

__host__ void test_device_mem_leak() {
    const unsigned int cycles = 1000;
    vector_t vec = new_vector_pinned(1000, RAND_UNIFORM);
    matrix_t mat = new_matrix_pinned(1000, 1000, RAND_UNIFORM);
    model_t mdl = new_model_pinned(100, 20, RAND_UNIFORM);
    printf("Device memory leak stress test: running %d cycles...\n", cycles);

    for (unsigned int i=0; i<cycles; i++) {
        vector_t d_vec = h2d_vector(vec);
        vector_t h_vec = d2h_vector(d_vec);
        assert_equal_vector(vec, h_vec);
        free_vector_pinned(h_vec);
        device_free_vector(d_vec);

        matrix_t d_mat = h2d_matrix(mat);
        matrix_t h_mat = d2h_matrix(d_mat);
        assert_equal_matrix(mat, h_mat);
        free_matrix_pinned(h_mat);
        device_free_matrix(d_mat);
        
        model_t d_mdl = h2d_model(mdl);
        device_free_model(d_mdl);
        printf("%d  ", i); fflush(stdout);
    }

    free_vector_pinned(vec);
    free_matrix_pinned(mat);
    free_model_pinned(mdl);
    printf("\n");
}