#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../utils/hpc.h"

#define ASSERT_NO_ERR(exp) {cudaSafeCall((exp)); cudaCheckError();}

__global__ void add( int *a, int *b, int *c )
{
    *c = *a + *b;
}

matrix_t cuda1_forward_mlp(matrix_t input_batch, model_t model)
{
    vector_t d_vec = h2d_vector(model->bias_list[0]);
    vector_t h_vec = d2h_vector(d_vec);
    print_vector(model->bias_list[0]);
    print_vector(h_vec);
    free_vector(h_vec);
    device_free_vector(d_vec);
    printf("vec OK\n");
    matrix_t d_mat = h2d_matrix(model->weights_list[0]);
    matrix_t h_mat = d2h_matrix(d_mat);
    print_matrix(model->weights_list[0]);
    print_matrix(h_mat);
    free_matrix(h_mat);
    device_free_matrix(d_mat);
    printf("mat OK\n");

    int a, b, c;	          /* host copies of a, b, c */ 
    int *d_a, *d_b, *d_c;	  /* device copies of a, b, c */
    const size_t size = sizeof(int);
    /* Allocate space for device copies of a, b, c */
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    /* Setup input values */
    a = 2; b = 7;
    /* Copy inputs to device */
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    /* Launch add() kernel on GPU */
    add<<<1,1>>>(d_a, d_b, d_c);
    /* Copy result back to host */
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    /* check result */
    if ( c != a + b ) {
        fprintf(stderr, "Test FAILED: expected %d, got %d\n", a+b, c);
    } else {
        printf("Test OK\n");
    }
    /* Cleanup */
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

vector_t h2d_vector(vector_t vec) {
    vector_t d_vec;
    ASSERT_NO_ERR(hipMalloc(&d_vec, sizeof(vector_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_vec->len), &(vec->len), sizeof(int), hipMemcpyHostToDevice));
    float* d_data;
    ASSERT_NO_ERR(hipMalloc(&d_data, vec->len*sizeof(float)));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&d_vec->data, &d_data, sizeof(float*), hipMemcpyHostToDevice));
    return d_vec;
}

vector_t d2h_vector(vector_t d_vec) {
    vector_t vec = (vector_t) malloc(sizeof(vector_s));
    ASSERT_NO_ERR(hipMemcpy(vec, d_vec, sizeof(vector_s), hipMemcpyDeviceToHost));
    float* d_data = (float*) malloc(vec->len*sizeof(float));
    ASSERT_NO_ERR(hipMemcpy(d_data, vec->data, vec->len*sizeof(float), hipMemcpyDeviceToHost));
    vec->data = d_data;
    return vec;
}

void device_free_vector(vector_t d_vec) {
    float* d_data;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_vec->data, sizeof(float*), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_vec));
}

matrix_t h2d_matrix(matrix_t mat) {
    matrix_t d_mat;
    ASSERT_NO_ERR(hipMalloc(&d_mat, sizeof(matrix_s)));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->m), &(mat->m), sizeof(int), hipMemcpyHostToDevice));
    ASSERT_NO_ERR(hipMemcpy(&(d_mat->n), &(mat->n), sizeof(int), hipMemcpyHostToDevice));
    float** d_data;
    ASSERT_NO_ERR(hipMalloc(&d_data, mat->m*sizeof(float*)));
    for (unsigned int i=0; i<mat->m; i++) {
        float *d_row;
        ASSERT_NO_ERR(hipMalloc(&d_row, mat->n*sizeof(float)));
        ASSERT_NO_ERR(hipMemcpy(d_row, mat->data[i], mat->n*sizeof(float), hipMemcpyHostToDevice));
        ASSERT_NO_ERR(hipMemcpy(&d_data[i], &d_row, sizeof(float*), hipMemcpyHostToDevice));
    }
    ASSERT_NO_ERR(hipMemcpy(&d_mat->data, &d_data, sizeof(float**), hipMemcpyHostToDevice));
    return d_mat;
}

matrix_t d2h_matrix(matrix_t d_mat) {
    matrix_t mat = (matrix_t) malloc(sizeof(matrix_s));
    ASSERT_NO_ERR(hipMemcpy(mat, d_mat, sizeof(matrix_s), hipMemcpyDeviceToHost));
    float** d_data = (float**) malloc(mat->m*sizeof(float*));
    ASSERT_NO_ERR(hipMemcpy(d_data, mat->data, mat->m*sizeof(float*), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<mat->m; i++) {
        float* d_row = (float*) malloc(mat->n*sizeof(float));
        ASSERT_NO_ERR(hipMemcpy(d_row, d_data[i], mat->n*sizeof(float), hipMemcpyDeviceToHost));
        d_data[i] = d_row;
    }
    mat->data = d_data;
    return mat;
}

void device_free_matrix(matrix_t d_mat) {
    float** d_data;
    unsigned int m;
    ASSERT_NO_ERR(hipMemcpy(&d_data, &d_mat->data, sizeof(float**), hipMemcpyDeviceToHost));
    ASSERT_NO_ERR(hipMemcpy(&m, &d_mat->m, sizeof(unsigned int), hipMemcpyDeviceToHost));
    for (unsigned int i=0; i<m; i++) {
        float* d_row;
        ASSERT_NO_ERR(hipMemcpy(&d_row, &d_data[i], sizeof(float*), hipMemcpyDeviceToHost));
        ASSERT_NO_ERR(hipFree(d_row));
    }
    ASSERT_NO_ERR(hipFree(d_data));
    ASSERT_NO_ERR(hipFree(d_mat));
}